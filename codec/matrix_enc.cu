#include "hip/hip_runtime.h"
/*
* Copyright (c) 2011 Apple Inc. All rights reserved.
*
* @APPLE_APACHE_LICENSE_HEADER_START@
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*
* @APPLE_APACHE_LICENSE_HEADER_END@
*/

/*
File:		matrix_enc.c

Contains:	ALAC mixing/matrixing encode routines.

Copyright:	(c) 2004-2011 Apple, Inc.
*/

#include <stdio.h>
#include <stdlib.h>

#include "matrixlib.h"
#include "ALACAudioTypes.h"

#define SIZE 512

// up to 24-bit "offset" macros for the individual bytes of a 20/24-bit word
#if TARGET_RT_BIG_ENDIAN
#define LBYTE	2
#define MBYTE	1
#define HBYTE	0
#else
#define LBYTE	0
#define MBYTE	1
#define HBYTE	2
#endif

/*
There is no plain middle-side option; instead there are various mixing
modes including middle-side, each lossless, as embodied in the mix()
and unmix() functions.  These functions exploit a generalized middle-side
transformation:

u := [(rL + (m-r)R)/m];
v := L - R;

where [ ] denotes integer floor.  The (lossless) inverse is

L = u + v - [rV/m];
R = L - v;
*/

// 16-bit routines

__global__ void gpu_mix16_1(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t m2, int32_t mixbits, int32_t mixres)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		int32_t		l, r;
		ip += stride * z;
		l = (int32_t)ip[0];
		r = (int32_t)ip[1];
		u[z] = (mixres * l + m2 * r) >> mixbits;
		v[z] = l - r;
	}
}

__global__ void gpu_mix16_2(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		ip += stride * z;
		u[z] = (int32_t)ip[0];
		v[z] = (int32_t)ip[1];
	}
}

__global__ void gpu_mix20_2(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		int32_t	l, r;

		l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		u[z] = (l << 8) >> 12;
		ip += 3 * z;

		r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		v[z] = (r << 8) >> 12;
		ip += (stride - 1) * 3 * z;
	}
}

void mix16(int16_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t mixbits, int32_t mixres)
{
	int16_t	*	ip = in;
	//	int32_t			j;

	int32_t *d_u, *d_v;
	int16_t *d_ip;

	hipMalloc(&d_u, numSamples * sizeof(int32_t));
	hipMalloc(&d_v, numSamples * sizeof(int32_t));
	hipMalloc(&d_ip, stride * numSamples * sizeof(int16_t));

	hipMemcpy(d_u, u, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_v, v, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_ip, ip, stride * numSamples * sizeof(int16_t), hipMemcpyHostToDevice);

	if (mixres != 0)
	{
//		printf("\nENTERS mix16 1\n");
		int32_t		mod = 1 << mixbits;
		int32_t		m2;

		/* matrixed stereo */
		m2 = mod - mixres;
		/*		for (j = 0; j < numSamples; j++)
		{
		int32_t		l, r;

		l = (int32_t)ip[0];
		r = (int32_t)ip[1];
		ip += stride;
		u[j] = (mixres * l + m2 * r) >> mixbits;
		v[j] = l - r;
		}*/
		gpu_mix16_1 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples, m2, mixbits, mixres);
	}
	else
	{
		/* Conventional separated stereo. */

		/*		printf("\n\n---------NEW---------\n\nNumber of Samples: %d\n", numSamples);

		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}

		printf("\n\n---------AFTER---------\n\n");*/


		gpu_mix16_2 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples);



		/*		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}*/




		/*		printf("\n\n---------NEW---------\n\nNumber of Samples: %d\n", numSamples);

		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}

		printf("\n\n---------AFTER---------\n\n");*/

		/*int32_t *du = (int32_t *)malloc(numSamples * sizeof(int32_t));
		int32_t *dv = (int32_t *)malloc(numSamples * sizeof(int32_t));
		memcpy(du, u, numSamples * sizeof(int32_t));
		memcpy(dv, v, numSamples * sizeof(int32_t));
		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n", u, v, u[0], v[0], u[numSamples], v[numSamples]);
		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n\n", du, dv, du[0], dv[0], du[numSamples], dv[numSamples]);
		free(du);
		free(dv);*/

		/*int16_t *dip = (int16_t *)malloc(numSamples * sizeof(int16_t) * 2);
		memcpy(dip, ip, 2 * numSamples * sizeof(int16_t));
		printf("%x \t\t %x \t\t %d \t\t %d \n", ip, ip + 2, ip[0], (ip + 2)[0]);
		printf("%x \t\t %x \t\t %d \t\t %d \n\n", dip, dip + 2, dip[0], (dip + 2)[0]);

		printf("%x \t\t %x \t\t %d \t\t %d \n", ip + (numSamples - 1) * stride, (ip + 2) + (numSamples - 1) * stride, (ip + (numSamples - 1) * stride)[0], (ip + 2 + (numSamples - 1) * stride)[0]);
		printf("%x \t\t %x \t\t %d \t\t %d \n\n", dip + (numSamples - 1) * stride, (dip + 2) + (numSamples - 1) * stride, (dip + (numSamples - 1) * stride)[0], (dip + 2 + (numSamples - 1) * stride)[0]);
		free(dip);*/

		/*for ( j = 0; j < numSamples; j++ )
		{
		u[j] = (int32_t) ip[0];
		v[j] = (int32_t) ip[1];
		ip += stride;
		}*/
		//		printf("%x \t\t %x \t\t %d \t\t %d \n\n\n", ip - 2, ip, (ip-2)[0], ip[0]);
		//		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n\n\n", u, v, u[0], v[0], u[numSamples], v[numSamples]);
		/*		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}*/
	}

	hipMemcpy(u, d_u, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
	hipMemcpy(v, d_v, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
	hipMemcpy(ip, d_ip, stride * numSamples * sizeof(int16_t), hipMemcpyDeviceToHost);

	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_ip);
}

// 20-bit routines
// - the 20 bits of data are left-justified in 3 bytes of storage but right-aligned for input/output predictor buffers

void mix20(uint8_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t mixbits, int32_t mixres)
{
	int32_t		l, r;
	uint8_t *	ip = in;
	int32_t			j;


	if (mixres != 0)
	{
		printf("\nENTERS mix20 1\n");
		/* matrixed stereo */
		int32_t		mod = 1 << mixbits;
		int32_t		m2 = mod - mixres;
		printf("\nENTERS mix20 1\n");
		for (j = 0; j < numSamples; j++)
		{
			l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			l = (l << 8) >> 12;
			ip += 3;

			r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			r = (r << 8) >> 12;
			ip += (stride - 1) * 3;

			u[j] = (mixres * l + m2 * r) >> mixbits;
			v[j] = l - r;
		}
	}
	else
	{

		/*int32_t *d_u, *d_v;
		int16_t *d_ip;

		hipMalloc(&d_u, numSamples * sizeof(int32_t));
		hipMalloc(&d_v, numSamples * sizeof(int32_t));
		hipMalloc(&d_ip, stride * numSamples * sizeof(int16_t));

		hipMemcpy(d_u, u, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_v, v, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_ip, ip, (stride-1) * 3 * numSamples * sizeof(int16_t), hipMemcpyHostToDevice);

		gpu_mix20_2 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples);*/

//		printf("\nENTERS mix20 2\n");

		/* Conventional separated stereo. */
		for (j = 0; j < numSamples; j++)
		{
			l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			u[j] = (l << 8) >> 12;
			ip += 3;

			r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			v[j] = (r << 8) >> 12;
			ip += (stride - 1) * 3;
		}

		/*hipMemcpy(u, d_u, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
		hipMemcpy(v, d_v, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
		hipMemcpy(ip, d_ip, stride * numSamples * sizeof(int16_t), hipMemcpyDeviceToHost);

		hipFree(d_u);
		hipFree(d_v);
		hipFree(d_ip);*/

	}
}

// 24-bit routines
// - the 24 bits of data are right-justified in the input/output predictor buffers

void mix24(uint8_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples,
	int32_t mixbits, int32_t mixres, uint16_t * shiftUV, int32_t bytesShifted)
{
	int32_t		l, r;
	uint8_t *	ip = in;
	int32_t			shift = bytesShifted * 8;
	uint32_t	mask = (1ul << shift) - 1;
	int32_t			j, k;

	if (mixres != 0)
	{
		/* matrixed stereo */
		int32_t		mod = 1 << mixbits;
		int32_t		m2 = mod - mixres;

		if (bytesShifted != 0)
		{
//			printf("\nENTERS mix24 1\n");
			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = (mixres * l + m2 * r) >> mixbits;
				v[j] = l - r;
			}
		}
		else
		{
			
			for (j = 0; j < numSamples; j++)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				u[j] = (mixres * l + m2 * r) >> mixbits;
				v[j] = l - r;
			}
		}
	}
	else
	{
		/* Conventional separated stereo. */
//		printf("\nENTERS mix24 2\n");
		if (bytesShifted != 0)
		{
			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = l;
				v[j] = r;
			}
		}
		else
		{
			for (j = 0; j < numSamples; j++)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				u[j] = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				v[j] = (r << 8) >> 8;
				ip += (stride - 1) * 3;
			}
		}
	}
}

// 32-bit routines
// - note that these really expect the internal data width to be < 32 but the arrays are 32-bit
// - otherwise, the calculations might overflow into the 33rd bit and be lost
// - therefore, these routines deal with the specified "unused lower" bytes in the "shift" buffers

void mix32(int32_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples,
	int32_t mixbits, int32_t mixres, uint16_t * shiftUV, int32_t bytesShifted)
{
	int32_t	*	ip = in;
	int32_t			shift = bytesShifted * 8;
	uint32_t	mask = (1ul << shift) - 1;
	int32_t		l, r;
	int32_t			j, k;

	if (mixres != 0)
	{
//		printf("\nENTERS mix32 1\n");
		int32_t		mod = 1 << mixbits;
		int32_t		m2;

		//Assert( bytesShifted != 0 );

		/* matrixed stereo with shift */
		m2 = mod - mixres;
		for (j = 0, k = 0; j < numSamples; j++, k += 2)
		{
			l = ip[0];
			r = ip[1];
			ip += stride;

			shiftUV[k + 0] = (uint16_t)(l & mask);
			shiftUV[k + 1] = (uint16_t)(r & mask);

			l >>= shift;
			r >>= shift;

			u[j] = (mixres * l + m2 * r) >> mixbits;
			v[j] = l - r;
		}
	}
	else
	{
//		printf("\nENTERS mix32 2\n");
		if (bytesShifted == 0)
		{
			/* de-interleaving w/o shift */
			for (j = 0; j < numSamples; j++)
			{
				u[j] = ip[0];
				v[j] = ip[1];
				ip += stride;
			}
		}
		else
		{
			/* de-interleaving with shift */
			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = ip[0];
				r = ip[1];
				ip += stride;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = l;
				v[j] = r;
			}
		}
	}
}

// 20/24-bit <-> 32-bit helper routines (not really matrixing but convenient to put here)

void copy20ToPredictor(uint8_t * in, uint32_t stride, int32_t * out, int32_t numSamples)
{
	uint8_t *	ip = in;
	int32_t			j;
	for (j = 0; j < numSamples; j++)
	{
		int32_t			val;

		// 20-bit values are left-aligned in the 24-bit input buffer but right-aligned in the 32-bit output buffer
		val = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		out[j] = (val << 8) >> 12;
		ip += stride * 3;
	}
}

void copy24ToPredictor(uint8_t * in, uint32_t stride, int32_t * out, int32_t numSamples)
{
	uint8_t *	ip = in;
	int32_t			j;
	for (j = 0; j < numSamples; j++)
	{
		int32_t			val;

		val = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		out[j] = (val << 8) >> 8;
		ip += stride * 3;
	}
}
