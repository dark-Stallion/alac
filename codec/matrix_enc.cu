#include "hip/hip_runtime.h"
/*
* Copyright (c) 2011 Apple Inc. All rights reserved.
*
* @APPLE_APACHE_LICENSE_HEADER_START@
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*
* @APPLE_APACHE_LICENSE_HEADER_END@
*/

/*
File:		matrix_enc.c

Contains:	ALAC mixing/matrixing encode routines.

Copyright:	(c) 2004-2011 Apple, Inc.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime_api.h>

#include "math.h"
#include "matrixlib.h"
#include "ALACAudioTypes.h"

#define SIZE 1024

// up to 24-bit "offset" macros for the individual bytes of a 20/24-bit word
#if TARGET_RT_BIG_ENDIAN
#define LBYTE	2
#define MBYTE	1
#define HBYTE	0
#else
#define LBYTE	0
#define MBYTE	1
#define HBYTE	2
#endif

/*
There is no plain middle-side option; instead there are various mixing
modes including middle-side, each lossless, as embodied in the mix()
and unmix() functions.  These functions exploit a generalized middle-side
transformation:

u := [(rL + (m-r)R)/m];
v := L - R;

where [ ] denotes integer floor.  The (lossless) inverse is

L = u + v - [rV/m];
R = L - v;
*/

// 16-bit routines

__global__ void gpu_mix16_1(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t m2, int32_t mixbits, int32_t mixres)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		int32_t		l, r;
		ip += stride * z;
		l = (int32_t)ip[0];
		r = (int32_t)ip[1];
		u[z] = (mixres * l + m2 * r) >> mixbits;
		v[z] = l - r;
	}
}

__global__ void gpu_mix16_2(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		ip += stride * z;
		u[z] = (int32_t)ip[0];
		v[z] = (int32_t)ip[1];
	}
}

__global__ void gpu_mix20_2(int16_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		int32_t	l, r;

		l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		u[z] = (l << 8) >> 12;
		ip += 3 * z;

		r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		v[z] = (r << 8) >> 12;
		ip += (stride - 1) * 3 * z;
	}
}

void mix16(int16_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t mixbits, int32_t mixres)
{
	int16_t	*ip = in;
	int32_t	j;


	if (mixres != 0)
	{
//		printf("\nENTERS mix16 1\n");
		int32_t		mod = 1 << mixbits;
		int32_t		m2;

		/* matrixed stereo */
		m2 = mod - mixres;

		/*double dtime = omp_get_wtime();*/

		for (j = 0; j < numSamples; j++)
		{
		int32_t	l, r;

		l = (int32_t)ip[0];
		r = (int32_t)ip[1];
		ip += stride;
		u[j] = (mixres * l + m2 * r) >> mixbits;
		v[j] = l - r;
		}

		/*dtime = omp_get_wtime() - dtime;
		printf("time in us %f\n", dtime * 1000000);*/


//		int32_t *d_u, *d_v;
//		int16_t *d_ip;
//
//		hipMalloc(&d_u, numSamples * sizeof(int32_t));
//		hipMalloc(&d_v, numSamples * sizeof(int32_t));
//		hipMalloc(&d_ip, stride * numSamples * sizeof(int16_t));
//
////		hipProfilerStart();
//
//		hipMemcpy(d_u, u, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
//		hipMemcpy(d_v, v, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
//		hipMemcpy(d_ip, ip, stride * numSamples * sizeof(int16_t), hipMemcpyHostToDevice);
//
//		//float elapsedTime;
//		//hipEvent_t start, stop;
//		//hipEventCreate(&start);
//		//hipEventCreate(&stop);
//		//hipEventRecord(start, 0);
//
//		gpu_mix16_1 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples, m2, mixbits, mixres);
//	
//		//hipEventRecord(stop, 0);
//		//hipEventSynchronize(stop);
//		//hipEventElapsedTime(&elapsedTime, start, stop);
//		//hipEventDestroy(start);
//		//hipEventDestroy(stop);
//		//printf("GPU Time elapsed: %f ms\n", elapsedTime);
//
//		hipMemcpy(u, d_u, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
//		hipMemcpy(v, d_v, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
//		hipMemcpy(ip, d_ip, stride * numSamples * sizeof(int16_t), hipMemcpyDeviceToHost);
//
////		hipProfilerStop();
//
//		hipFree(d_u);
//		hipFree(d_v);
//		hipFree(d_ip);
	
	}
	else
	{
		/* Conventional separated stereo. */

		/*		printf("\n\n---------NEW---------\n\nNumber of Samples: %d\n", numSamples);

		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}

		printf("\n\n---------AFTER---------\n\n");*/


//		gpu_mix16_2<<< (numSamples + SIZE - 1) / SIZE, SIZE >>>(d_ip, stride, d_u, d_v, numSamples);



		/*		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}*/




		/*		printf("\n\n---------NEW---------\n\nNumber of Samples: %d\n", numSamples);

		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}

		printf("\n\n---------AFTER---------\n\n");*/

		/*int32_t *du = (int32_t *)malloc(numSamples * sizeof(int32_t));
		int32_t *dv = (int32_t *)malloc(numSamples * sizeof(int32_t));
		memcpy(du, u, numSamples * sizeof(int32_t));
		memcpy(dv, v, numSamples * sizeof(int32_t));
		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n", u, v, u[0], v[0], u[numSamples], v[numSamples]);
		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n\n", du, dv, du[0], dv[0], du[numSamples], dv[numSamples]);
		free(du);
		free(dv);*/

		/*int16_t *dip = (int16_t *)malloc(numSamples * sizeof(int16_t) * 2);
		memcpy(dip, ip, 2 * numSamples * sizeof(int16_t));
		printf("%x \t\t %x \t\t %d \t\t %d \n", ip, ip + 2, ip[0], (ip + 2)[0]);
		printf("%x \t\t %x \t\t %d \t\t %d \n\n", dip, dip + 2, dip[0], (dip + 2)[0]);

		printf("%x \t\t %x \t\t %d \t\t %d \n", ip + (numSamples - 1) * stride, (ip + 2) + (numSamples - 1) * stride, (ip + (numSamples - 1) * stride)[0], (ip + 2 + (numSamples - 1) * stride)[0]);
		printf("%x \t\t %x \t\t %d \t\t %d \n\n", dip + (numSamples - 1) * stride, (dip + 2) + (numSamples - 1) * stride, (dip + (numSamples - 1) * stride)[0], (dip + 2 + (numSamples - 1) * stride)[0]);
		free(dip);*/

		for ( j = 0; j < numSamples; j++ )
		{
			u[j] = (int32_t) ip[0];
			v[j] = (int32_t) ip[1];
			ip += stride;
		}
		//		printf("%x \t\t %x \t\t %d \t\t %d \n\n\n", ip - 2, ip, (ip-2)[0], ip[0]);
		//		printf("%x \t\t %x \t\t %d \t\t %d \t\t %d \t\t %d \n\n\n", u, v, u[0], v[0], u[numSamples], v[numSamples]);
		/*		for (int i = 0; i < 10; i++){
		printf("%x\t\t%d\t\t%d\t\t\t%d\t\t%d\n", ip + stride*i, ip[0], ip[1], u[i], v[i]);
		}*/
	}

}

// 20-bit routines
// - the 20 bits of data are left-justified in 3 bytes of storage but right-aligned for input/output predictor buffers

void mix20(uint8_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t mixbits, int32_t mixres)
{
	int32_t		l, r;
	uint8_t *	ip = in;
	int32_t			j;


	if (mixres != 0)
	{
		printf("\nENTERS mix20 1\n");
		/* matrixed stereo */
		int32_t		mod = 1 << mixbits;
		int32_t		m2 = mod - mixres;
		for (j = 0; j < numSamples; j++)
		{
			l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			l = (l << 8) >> 12;
			ip += 3;

			r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			r = (r << 8) >> 12;
			ip += (stride - 1) * 3;

			u[j] = (mixres * l + m2 * r) >> mixbits;
			v[j] = l - r;
		}
	}
	else
	{

		/*int32_t *d_u, *d_v;
		int16_t *d_ip;

		hipMalloc(&d_u, numSamples * sizeof(int32_t));
		hipMalloc(&d_v, numSamples * sizeof(int32_t));
		hipMalloc(&d_ip, stride * numSamples * sizeof(int16_t));

		hipMemcpy(d_u, u, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_v, v, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_ip, ip, (stride-1) * 3 * numSamples * sizeof(int16_t), hipMemcpyHostToDevice);

		gpu_mix20_2 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples);*/

		printf("\nENTERS mix20 2\n");

		/* Conventional separated stereo. */
		for (j = 0; j < numSamples; j++)
		{
			l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			u[j] = (l << 8) >> 12;
			ip += 3;

			r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
			v[j] = (r << 8) >> 12;
			ip += (stride - 1) * 3;
		}

		/*hipMemcpy(u, d_u, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
		hipMemcpy(v, d_v, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
		hipMemcpy(ip, d_ip, stride * numSamples * sizeof(int16_t), hipMemcpyDeviceToHost);

		hipFree(d_u);
		hipFree(d_v);
		hipFree(d_ip);*/

	}
}

// 24-bit routines
// - the 24 bits of data are right-justified in the input/output predictor buffers

__global__ void gpu_mix24_1(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, int32_t mixres, uint32_t mask, int32_t m2, int32_t mixbits, int32_t shift)
{
	int z = threadIdx.x + blockIdx.x * blockDim.x;
	if (z < numSamples)
	{
		int32_t		l, r;
		int32_t k = z * 2;

		int32_t x1 = 3 * z;
		int32_t y1 = (stride - 1) * 3 * z;

		int32_t x2 = 3 * (z+1);

		l = (int32_t)(((uint32_t)(ip + x1 + y1)[HBYTE] << 16) | ((uint32_t)(ip + x1 + y1)[MBYTE] << 8) | (uint32_t)(ip + x1 + y1)[LBYTE]);
		l = (l << 8) >> 8;

		r = (int32_t)(((uint32_t)(ip + x2 + y1)[HBYTE] << 16) | ((uint32_t)(ip + x2 + y1)[MBYTE] << 8) | (uint32_t)(ip + x2 + y1)[LBYTE]);
		r = (r << 8) >> 8;

		shiftUV[k + 0] = (uint16_t)(l & mask);
		shiftUV[k + 1] = (uint16_t)(r & mask);

		l >>= shift;
		r >>= shift;

		u[z] = (mixres * l + m2 * r) >> mixbits;
		v[z] = l - r;
	}
}

void mix24(uint8_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples,
	int32_t mixbits, int32_t mixres, uint16_t * shiftUV, int32_t bytesShifted)
{
	int32_t		l, r;
	uint8_t *	ip = in;
	int32_t			shift = bytesShifted * 8;
	uint32_t	mask = (1ul << shift) - 1;
	int32_t			j, k;

	if (mixres != 0)
	{
		/* matrixed stereo */
		int32_t		mod = 1 << mixbits;
		int32_t		m2 = mod - mixres;

		if (bytesShifted != 0)
		{


			/*int32_t *d_u, *d_v;
			uint8_t *d_ip;
			uint16_t *d_shiftUV;

			hipMalloc(&d_u, numSamples * sizeof(int32_t));
			hipMalloc(&d_v, numSamples * sizeof(int32_t));
			hipMalloc(&d_ip, (stride - 1) * 3 * numSamples * sizeof(uint8_t));
			hipMalloc(&d_shiftUV, 2 * numSamples * sizeof(uint16_t));
			
			hipMemcpy(d_u, u, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
			hipMemcpy(d_v, v, numSamples * sizeof(int32_t), hipMemcpyHostToDevice);
			hipMemcpy(d_ip, ip, (stride - 1) * 3 * numSamples * sizeof(uint8_t), hipMemcpyHostToDevice);
			hipMemcpy(d_shiftUV, shiftUV, 2 * numSamples * sizeof(uint16_t), hipMemcpyHostToDevice);
			
			gpu_mix24_1 << < (numSamples + SIZE - 1) / SIZE, SIZE >> >(d_ip, stride, d_u, d_v, numSamples, d_shiftUV, mixres, mask, m2, mixbits, shift);
			
			hipMemcpy(u, d_u, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
			hipMemcpy(v, d_v, numSamples * sizeof(int32_t), hipMemcpyDeviceToHost);
			hipMemcpy(ip, d_ip, (stride - 1) * 3 * numSamples * sizeof(uint8_t), hipMemcpyDeviceToHost);
			hipMemcpy(shiftUV, d_shiftUV, 2 * numSamples * sizeof(uint16_t), hipMemcpyDeviceToHost);
			
			hipFree(d_u);
			hipFree(d_v);
			hipFree(d_ip);
			hipFree(d_shiftUV);*/


			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = (mixres * l + m2 * r) >> mixbits;
				v[j] = l - r;
			}
		}
		else
		{
			
			for (j = 0; j < numSamples; j++)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				u[j] = (mixres * l + m2 * r) >> mixbits;
				v[j] = l - r;
			}
		}
	}
	else
	{
		/* Conventional separated stereo. */
		if (bytesShifted != 0)
		{
			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				l = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				r = (r << 8) >> 8;
				ip += (stride - 1) * 3;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = l;
				v[j] = r;
			}
		}
		else
		{
			for (j = 0; j < numSamples; j++)
			{
				l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				u[j] = (l << 8) >> 8;
				ip += 3;

				r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
				v[j] = (r << 8) >> 8;
				ip += (stride - 1) * 3;
			}
		}
	}
}

// 32-bit routines
// - note that these really expect the internal data width to be < 32 but the arrays are 32-bit
// - otherwise, the calculations might overflow into the 33rd bit and be lost
// - therefore, these routines deal with the specified "unused lower" bytes in the "shift" buffers

void mix32(int32_t * in, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples,
	int32_t mixbits, int32_t mixres, uint16_t * shiftUV, int32_t bytesShifted)
{
	int32_t	*	ip = in;
	int32_t			shift = bytesShifted * 8;
	uint32_t	mask = (1ul << shift) - 1;
	int32_t		l, r;
	int32_t			j, k;

	if (mixres != 0)
	{
		printf("\nENTERS mix32 1\n");
		int32_t		mod = 1 << mixbits;
		int32_t		m2;

		//Assert( bytesShifted != 0 );

		/* matrixed stereo with shift */
		m2 = mod - mixres;
		for (j = 0, k = 0; j < numSamples; j++, k += 2)
		{
			l = ip[0];
			r = ip[1];
			ip += stride;

			shiftUV[k + 0] = (uint16_t)(l & mask);
			shiftUV[k + 1] = (uint16_t)(r & mask);

			l >>= shift;
			r >>= shift;

			u[j] = (mixres * l + m2 * r) >> mixbits;
			v[j] = l - r;
		}
	}
	else
	{
		printf("\nENTERS mix32 2\n");
		if (bytesShifted == 0)
		{
			/* de-interleaving w/o shift */
			for (j = 0; j < numSamples; j++)
			{
				u[j] = ip[0];
				v[j] = ip[1];
				ip += stride;
			}
		}
		else
		{
			/* de-interleaving with shift */
			for (j = 0, k = 0; j < numSamples; j++, k += 2)
			{
				l = ip[0];
				r = ip[1];
				ip += stride;

				shiftUV[k + 0] = (uint16_t)(l & mask);
				shiftUV[k + 1] = (uint16_t)(r & mask);

				l >>= shift;
				r >>= shift;

				u[j] = l;
				v[j] = r;
			}
		}
	}
}

// 20/24-bit <-> 32-bit helper routines (not really matrixing but convenient to put here)

void copy20ToPredictor(uint8_t * in, uint32_t stride, int32_t * out, int32_t numSamples)
{
	printf("\nENTERS copy20ToPredictor\n");
	uint8_t *	ip = in;
	int32_t			j;
	for (j = 0; j < numSamples; j++)
	{
		int32_t			val;

		// 20-bit values are left-aligned in the 24-bit input buffer but right-aligned in the 32-bit output buffer
		val = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		out[j] = (val << 8) >> 12;
		ip += stride * 3;
	}
}

void copy24ToPredictor(uint8_t * in, uint32_t stride, int32_t * out, int32_t numSamples)
{
	printf("\nENTERS copy24ToPredictor\n");
	uint8_t *	ip = in;
	int32_t			j;
	for (j = 0; j < numSamples; j++)
	{
		int32_t			val;

		val = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
		out[j] = (val << 8) >> 8;
		ip += stride * 3;
	}
}
