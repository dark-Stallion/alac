#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011 Apple Inc. All rights reserved.
 *
 * @APPLE_APACHE_LICENSE_HEADER_START@
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 * 
 * @APPLE_APACHE_LICENSE_HEADER_END@
 */

/*
	File:		ALACDecoder.cpp
*/

#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "ALACDecoder.h"

#include "dplib.h"
#include "aglib.h"
#include "matrixlib.h"

#include "ALACBitUtilities.h"
#include "EndianPortable.h"

#define SIZE 1024

// up to 24-bit "offset" macros for the individual bytes of a 20/24-bit word
#if TARGET_RT_BIG_ENDIAN
#define LBYTE	2
#define MBYTE	1
#define HBYTE	0
#else
#define LBYTE	0
#define MBYTE	1
#define HBYTE	2
#endif

// constants/data
const uint32_t kMaxBitDepth = 32;			// max allowed bit depth is 32


// prototypes
static void Zero16( int16_t * buffer, uint32_t numItems, uint32_t stride );
static void Zero24( uint8_t * buffer, uint32_t numItems, uint32_t stride );
static void Zero32( int32_t * buffer, uint32_t numItems, uint32_t stride );

/*
	Constructor
*/
ALACDecoder::ALACDecoder() :
	mMixBufferU( nil ),
	mMixBufferV( nil ),
	mPredictor( nil ),
	mShiftBuffer( nil )
{
	memset( &mConfig, 0, sizeof(mConfig) );
}

/*
	Destructor
*/
ALACDecoder::~ALACDecoder()
{
	// delete the matrix mixing buffers
	if ( mMixBufferU )
    {
		free(mMixBufferU);
		hipFree(d_u);
        mMixBufferU = NULL;
    }
	if ( mMixBufferV )
    {
		free(mMixBufferV);
		hipFree(d_v);
        mMixBufferV = NULL;
    }
	
	// delete the dynamic predictor's "corrector" buffer
	// - note: mShiftBuffer shares memory with this buffer
	if ( mPredictor )
    {
		free(mPredictor);
		hipFree(d_shiftUV);
        mPredictor = NULL;
    }
}

/*
	Init()
	- initialize the decoder with the given configuration
*/
int32_t ALACDecoder::Init( void * inMagicCookie, uint32_t inMagicCookieSize, int X )
{
	int32_t		status = ALAC_noErr;
    ALACSpecificConfig theConfig;
    uint8_t * theActualCookie = (uint8_t *)inMagicCookie;
    uint32_t theCookieBytesRemaining = inMagicCookieSize;

    // For historical reasons the decoder needs to be resilient to magic cookies vended by older encoders.
    // As specified in the ALACMagicCookieDescription.txt document, there may be additional data encapsulating 
    // the ALACSpecificConfig. This would consist of format ('frma') and 'alac' atoms which precede the
    // ALACSpecificConfig. 
    // See ALACMagicCookieDescription.txt for additional documentation concerning the 'magic cookie'
    
    // skip format ('frma') atom if present
    if (theActualCookie[4] == 'f' && theActualCookie[5] == 'r' && theActualCookie[6] == 'm' && theActualCookie[7] == 'a')
    {
        theActualCookie += 12;
        theCookieBytesRemaining -= 12;
    }
    
    // skip 'alac' atom header if present
    if (theActualCookie[4] == 'a' && theActualCookie[5] == 'l' && theActualCookie[6] == 'a' && theActualCookie[7] == 'c')
    {
        theActualCookie += 12;
        theCookieBytesRemaining -= 12;
    }

    // read the ALACSpecificConfig
    if (theCookieBytesRemaining >= sizeof(ALACSpecificConfig))
    {
        theConfig.frameLength = Swap32BtoN(((ALACSpecificConfig *)theActualCookie)->frameLength);
        theConfig.compatibleVersion = ((ALACSpecificConfig *)theActualCookie)->compatibleVersion;
        theConfig.bitDepth = ((ALACSpecificConfig *)theActualCookie)->bitDepth;
        theConfig.pb = ((ALACSpecificConfig *)theActualCookie)->pb;
        theConfig.mb = ((ALACSpecificConfig *)theActualCookie)->mb;
        theConfig.kb = ((ALACSpecificConfig *)theActualCookie)->kb;
        theConfig.numChannels = ((ALACSpecificConfig *)theActualCookie)->numChannels;
        theConfig.maxRun = Swap16BtoN(((ALACSpecificConfig *)theActualCookie)->maxRun);
        theConfig.maxFrameBytes = Swap32BtoN(((ALACSpecificConfig *)theActualCookie)->maxFrameBytes);
        theConfig.avgBitRate = Swap32BtoN(((ALACSpecificConfig *)theActualCookie)->avgBitRate);
        theConfig.sampleRate = Swap32BtoN(((ALACSpecificConfig *)theActualCookie)->sampleRate);

        mConfig = theConfig;
        
        RequireAction( mConfig.compatibleVersion <= kALACVersion, return kALAC_ParamError; );

		total_numSamples = 0;

        // allocate mix buffers
        mMixBufferU = (int32_t *) calloc( mConfig.frameLength * sizeof(int32_t), 1 );
        mMixBufferV = (int32_t *) calloc( mConfig.frameLength * sizeof(int32_t), 1 );

        // allocate dynamic predictor buffer
        mPredictor = (int32_t *) calloc( mConfig.frameLength * sizeof(int32_t), 1 );

        // "shift off" buffer shares memory with predictor buffer
        mShiftBuffer = (uint16_t *) mPredictor;
        
		hipMalloc(&d_u, X * mConfig.frameLength * sizeof(int32_t));
		hipMalloc(&d_v, X * mConfig.frameLength * sizeof(int32_t));
		hipMalloc(&d_shiftUV, X * mConfig.frameLength * sizeof(int32_t));
		hipMalloc(&d_numSamples, X * sizeof(uint32_t));
		hipMalloc(&d_mixRes, X * sizeof(int32_t));
		hipMalloc(&d_mixBits, X * sizeof(int32_t));

        RequireAction( (mMixBufferU != nil) && (mMixBufferV != nil) && (mPredictor != nil),
                        status = kALAC_MemFullError; goto Exit; );
     }
    else
    {
        status = kALAC_ParamError;
    }

    // skip to Channel Layout Info
    // theActualCookie += sizeof(ALACSpecificConfig);
    
    // Currently, the Channel Layout Info portion of the magic cookie (as defined in the 
    // ALACMagicCookieDescription.txt document) is unused by the decoder. 
    
Exit:
	return status;
}


__global__ void gpu_unmix16(int32_t * u, int32_t * v, int16_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{

		int32_t		l, r;
		int16_t * op = out + (index * theOutputPacketBytes) / 2;

		if (mixres[index] != 0)
		{
			/* matrixed stereo */
			l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
			r = l - (v + index * frameLength)[z];

			op += stride* z;
			op[0] = (int16_t)l;
			op[1] = (int16_t)r;
		}
		else
		{
			/* Conventional separated stereo. */
			op += stride * z;
			op[0] = (int16_t)(u + index * frameLength)[z];
			op[1] = (int16_t)(v + index * frameLength)[z];
		}
	}
	/*if (block == 0 && index == 0 && z == 0)
		printf("%d\n", mixbits[8]);*/
}


__global__ void gpu_unmix20(int32_t * u, int32_t * v, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{

		int32_t		l, r;
		uint8_t * op = out + (index * theOutputPacketBytes) / 2;

		if (mixres[index] != 0)
		{
			/* matrixed stereo */
			l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
			r = l - (v + index * frameLength)[z];

			l <<= 4;
			r <<= 4;

			op += 3 * z;
			op += (stride - 1) * 3 * z;
			op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
			op += 3;

			op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
		}
		else
		{
			/* Conventional separated stereo. */
			int32_t		val;

			val = (u + index * frameLength)[z] << 4;
			op += 3 * z;
			op += (stride - 1) * 3 * z;
			op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
			op += 3;

			val = (v + index * frameLength)[z] << 4;
			op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
		}
	}
}

// 24-bit routines
// - the 24 bits of data are right-justified in the input/output predictor buffers

__global__ void gpu_unmix24(int32_t * u, int32_t * v, uint8_t * out, uint32_t stride, uint32_t * numSamples,
	int32_t * mixbits, int32_t * mixres, uint16_t * shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{

		int32_t			shift = bytesShifted * 8;
		int32_t		l, r;
		int32_t k = z * 2;
		uint8_t * op = out + (index * theOutputPacketBytes);

		if (mixres[index] != 0)
		{
			/* matrixed stereo */
			l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
			r = l - (v + index * frameLength)[z];

			l = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
			r = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

			op += 3 * z;
			op += (stride - 1) * 3 * z;
			op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
			op += 3;

			op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
		}
		else
		{
			/* Conventional separated stereo. */
			l = (u + index * frameLength)[z];
			r = (v + index * frameLength)[z];

			l = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
			r = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

			op += 3 * z;
			op += (stride - 1) * 3 * z;
			op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
			op += 3;

			op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
			op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
			op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
		}
	}
}


__global__ void gpu_unmix32(int32_t * u, int32_t * v, int32_t * out, uint32_t stride, uint32_t * numSamples,
	int32_t * mixbits, int32_t * mixres, uint16_t * shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;
	if (z < numSamples[index])
	{
		int32_t			shift = bytesShifted * 8;
		int32_t		l, r;
		int32_t k = z * 2;
		int32_t * op = out + (index * theOutputPacketBytes)/4;

		if (mixres[index] != 0)
		{
			//Assert( bytesShifted != 0 );

			/* matrixed stereo with shift */
			int32_t		lt, rt;

			lt = (u + index * frameLength)[z];
			rt = (v + index * frameLength)[z];

			l = lt + rt - ((mixres[index] * rt) >> mixbits[index]);
			r = l - rt;

			op += stride * z;
			op[0] = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
			op[1] = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];
		}
		else
		{
			/* interleaving with shift */
			op += stride * z;
			op[0] = ((u + index * frameLength)[z] << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
			op[1] = ((v + index * frameLength)[z] << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

		}
	}
}

__global__ void gpu_copyPredictorTo16(int32_t * in, int16_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		int16_t * op = out + (index * theOutputPacketBytes)/2;
		
		op[z] = (int16_t)(in + index * frameLength)[z];
	}
}

__global__ void gpu_copyPredictorTo20(int32_t * in, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		uint8_t * op = out + (index * theOutputPacketBytes);
		int32_t	val = (in + index * frameLength)[z];
		op += (stride * 3 * z);

		op[HBYTE] = (uint8_t)((val >> 12) & 0xffu);
		op[MBYTE] = (uint8_t)((val >> 4) & 0xffu);
		op[LBYTE] = (uint8_t)((val << 4) & 0xffu);
	}
}

__global__ void gpu_copyPredictorTo24(int32_t * in, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		uint8_t * op = out + (index * theOutputPacketBytes);
		int32_t	val = (in + index * frameLength)[z];
		op += (stride * 3 * z);

		op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
		op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
		op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
	}
}

__global__ void gpu_copyPredictorTo24Shift(int32_t * in, uint16_t * shift, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{

	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		uint8_t * op = out + (index * theOutputPacketBytes);
		int32_t	shiftVal = bytesShifted * 8;

		//Assert( bytesShifted != 0 );

		int32_t	val = (in + index * frameLength)[z];

		val = (val << shiftVal) | (uint32_t)(shift + index * frameLength * 2 )[z];


		op += (stride * 3 * z);
		op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
		op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
		op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
	}
}

__global__ void gpu_copyPredictorTo32(int32_t * in, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		int32_t * op = out + (index * theOutputPacketBytes)/4;
		op[stride * z] = (in + index * frameLength)[z];

	}
}

__global__ void gpu_copyPredictorTo32Shift(int32_t * in, uint16_t * shift, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{

	int block = blockIdx.x % 4;
	int index = blockIdx.x / 4;
	int z = threadIdx.x + block * blockDim.x;

	if (z < numSamples[index])
	{
		int32_t * op = out + (index * theOutputPacketBytes)/4;
		int32_t	shiftVal = bytesShifted * 8;

		//Assert( bytesShifted != 0 );

		op += stride * z;
		op[0] = ((in + index * frameLength)[z] << shiftVal) | (uint32_t)(shift + index * frameLength * 2)[z];

	}
}

void ALACDecoder::fillWriteBuffer(void * sampleBuffer, uint32_t numChannels, int32_t theOutputPacketBytes, int X){

	switch (tag){

		case ID_SCE:
		case ID_LFE:
		{
			switch (mConfig.bitDepth)
			{
				case 16:
					gpu_copyPredictorTo16 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, (int16_t *)sampleBuffer,
						numChannels, d_numSamples, theOutputPacketBytes, mConfig.frameLength);
					break;
				case 20:
					gpu_copyPredictorTo20 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, (uint8_t *)sampleBuffer,
						numChannels, d_numSamples, theOutputPacketBytes, mConfig.frameLength);
					break;
				case 24:
					if (bytesShifted != 0){
						gpu_copyPredictorTo24Shift << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_shiftUV, (uint8_t *)sampleBuffer,
							numChannels, d_numSamples, (int32_t)bytesShifted, theOutputPacketBytes, mConfig.frameLength);
					}
					else{
						gpu_copyPredictorTo24 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, (uint8_t *)sampleBuffer,
							numChannels, d_numSamples, theOutputPacketBytes, mConfig.frameLength);
					}
					break;
				case 32:
					if (bytesShifted != 0){
						gpu_copyPredictorTo32Shift << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_shiftUV, (int32_t *)sampleBuffer,
						numChannels, d_numSamples, (int32_t)bytesShifted, theOutputPacketBytes, mConfig.frameLength);
					}
					else{
						gpu_copyPredictorTo32 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, (int32_t *)sampleBuffer,
						numChannels, d_numSamples, theOutputPacketBytes, mConfig.frameLength);
					}
					break;
			}
			break;
		}
		case ID_CPE:
		{
			switch (mConfig.bitDepth)
			{
				case 16:
					gpu_unmix16 << < ((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_v, (int16_t *)sampleBuffer,
						numChannels, d_numSamples, d_mixBits, d_mixRes, theOutputPacketBytes, mConfig.frameLength);
					break;
				case 20:
					gpu_unmix20 << < ((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_v, (uint8_t *)sampleBuffer,
						numChannels, d_numSamples, d_mixBits, d_mixRes, theOutputPacketBytes, mConfig.frameLength);
					break;
				case 24:
					gpu_unmix24 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_v, (uint8_t *)sampleBuffer,
						numChannels, d_numSamples, d_mixBits, d_mixRes, d_shiftUV, (int32_t)bytesShifted, theOutputPacketBytes, mConfig.frameLength);
					//hipMemcpy(out24, d_out, numChannels * 3 * numSamples * sizeof(uint8_t), hipMemcpyDeviceToHost);
					break;
				case 32:
					gpu_unmix32 << <((total_numSamples + SIZE - 1) / SIZE), SIZE >> >(d_u, d_v, (int32_t *)sampleBuffer,
						numChannels, d_numSamples, d_mixBits, d_mixRes, d_shiftUV, (int32_t)bytesShifted, theOutputPacketBytes, mConfig.frameLength);
					break;
			}
			break;
		}

	}

}


/*
	Decode()
	- the decoded samples are interleaved into the output buffer in the order they arrive in
	  the bitstream
*/
int32_t ALACDecoder::Decode(BitBuffer * bits, uint32_t numSamples, uint32_t numChannels, uint32_t * outNumSamples, uint32_t mBytesPerFrame, int X)
{
	BitBuffer			shiftBits;
	uint32_t            bits1, bits2;
	uint8_t				elementInstanceTag;
	AGParamRec			agParams;
	uint32_t				channelIndex;
	int16_t				coefsU[32];		// max possible size is 32 although NUMCOEPAIRS is the current limit
	int16_t				coefsV[32];
	uint8_t				numU, numV;
	uint8_t				mixBits;
	int8_t				mixRes;
	uint16_t			unusedHeader;
	uint8_t				escapeFlag;
	uint32_t			chanBits;
	//uint8_t				bytesShifted;
	uint32_t			shift;
	uint8_t				modeU, modeV;
	uint32_t			denShiftU, denShiftV;
	uint16_t			pbFactorU, pbFactorV;
	uint16_t			pb;
	int16_t *			out16;
	uint8_t *			out20;
	uint8_t *			out24;
	int32_t *			out32;
	uint8_t				headerByte;
	uint8_t				partialFrame;
	uint32_t			extraBits;
	int32_t				val;
	uint32_t			i, j;
	int32_t             status;
	
	//RequireAction( (bits != nil) && (sampleBuffer != nil) && (outNumSamples != nil), return kALAC_ParamError; );
	RequireAction( numChannels > 0, return kALAC_ParamError; );

	mActiveElements = 0;
	channelIndex	= 0;

	status = ALAC_noErr;
	*outNumSamples = numSamples;

	while ( status == ALAC_noErr )
	{
		// bail if we ran off the end of the buffer
    	RequireAction( bits->cur < bits->end, status = kALAC_ParamError; goto Exit; );

		// copy global decode params for this element
		pb = mConfig.pb;

		// read element tag
		tag = BitBufferReadSmall( bits, 3 );
		switch ( tag )
		{
			case ID_SCE:
			case ID_LFE:
			{
				// mono/LFE channel
				elementInstanceTag = BitBufferReadSmall( bits, 4 );
				mActiveElements |= (1u << elementInstanceTag);

				// read the 12 unused header bits
				unusedHeader = (uint16_t) BitBufferRead( bits, 12 );
				RequireAction( unusedHeader == 0, status = kALAC_ParamError; goto Exit; );

				// read the 1-bit "partial frame" flag, 2-bit "shift-off" flag & 1-bit "escape" flag
				headerByte = (uint8_t) BitBufferRead( bits, 4 );
				
				partialFrame = headerByte >> 3;
				
				bytesShifted = (headerByte >> 1) & 0x3u;
				RequireAction( bytesShifted != 3, status = kALAC_ParamError; goto Exit; );

				shift = bytesShifted * 8;

				escapeFlag = headerByte & 0x1;

				chanBits = mConfig.bitDepth - (bytesShifted * 8);
				
				// check for partial frame to override requested numSamples
				if ( partialFrame != 0 )
				{
					numSamples  = BitBufferRead( bits, 16 ) << 16;
					numSamples |= BitBufferRead( bits, 16 );
				}

				if ( escapeFlag == 0 )
				{
					// compressed frame, read rest of parameters
					mixBits	= (uint8_t) BitBufferRead( bits, 8 );
					mixRes	= (int8_t) BitBufferRead( bits, 8 );
					//Assert( (mixBits == 0) && (mixRes == 0) );		// no mixing for mono

					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					modeU		= headerByte >> 4;
					denShiftU	= headerByte & 0xfu;
					
					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					pbFactorU	= headerByte >> 5;
					numU		= headerByte & 0x1fu;

					for ( i = 0; i < numU; i++ )
						coefsU[i] = (int16_t) BitBufferRead( bits, 16 );
					
					// if shift active, skip the the shift buffer but remember where it starts
					if ( bytesShifted != 0 )
					{
						shiftBits = *bits;
						BitBufferAdvance( bits, (bytesShifted * 8) * numSamples ); 
					}

					// decompress
					set_ag_params( &agParams, mConfig.mb, (pb * pbFactorU) / 4, mConfig.kb, numSamples, numSamples, mConfig.maxRun );
					status = dyn_decomp( &agParams, bits, mPredictor, numSamples, chanBits, &bits1 );
					RequireNoErr( status, goto Exit; );

					if ( modeU == 0 )
					{
						unpc_block( mPredictor, mMixBufferU, numSamples, &coefsU[0], numU, chanBits, denShiftU );
					}
					else
					{
						// the special "numActive == 31" mode can be done in-place
						unpc_block( mPredictor, mPredictor, numSamples, nil, 31, chanBits, 0 );
						unpc_block( mPredictor, mMixBufferU, numSamples, &coefsU[0], numU, chanBits, denShiftU );
					}
				}
				else
				{
					//Assert( bytesShifted == 0 );

					// uncompressed frame, copy data into the mix buffer to use common output code
					shift = 32 - chanBits;
					if ( chanBits <= 16 )
					{
						for ( i = 0; i < numSamples; i++ )
						{
							val = (int32_t) BitBufferRead( bits, (uint8_t) chanBits );
							val = (val << shift) >> shift;
							mMixBufferU[i] = val;
						}
					}
					else
					{
						// BitBufferRead() can't read more than 16 bits at a time so break up the reads
						extraBits = chanBits - 16;
						for ( i = 0; i < numSamples; i++ )
						{
							val = (int32_t) BitBufferRead( bits, 16 );
							val = (val << 16) >> shift;
							mMixBufferU[i] = val | BitBufferRead( bits, (uint8_t) extraBits );
						}
					}

					mixBits = mixRes = 0;
					bits1 = chanBits * numSamples;
					bytesShifted = 0;
				}

				// now read the shifted values into the shift buffer
				if ( bytesShifted != 0 )
				{
					shift = bytesShifted * 8;
					//Assert( shift <= 16 );

					for ( i = 0; i < numSamples; i++ )
						mShiftBuffer[i] = (uint16_t) BitBufferRead( &shiftBits, (uint8_t) shift );
				}

				// convert 32-bit integers into output buffer

				hipMemcpy(d_u + (X * mConfig.frameLength), mMixBufferU, mConfig.frameLength * sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_shiftUV + (X * mConfig.frameLength * 2), mShiftBuffer, mConfig.frameLength * sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_numSamples + X, &numSamples, sizeof(int32_t), hipMemcpyHostToDevice);

				total_numSamples += numSamples;

				channelIndex += 1;
				*outNumSamples = numSamples;
				break;
			}

			case ID_CPE:
			{
				// if decoding this pair would take us over the max channels limit, bail
				if ( (channelIndex + 2) > numChannels )
					goto NoMoreChannels;

				// stereo channel pair
				elementInstanceTag = BitBufferReadSmall( bits, 4 );
				mActiveElements |= (1u << elementInstanceTag);

				// read the 12 unused header bits
				unusedHeader = (uint16_t) BitBufferRead( bits, 12 );
				RequireAction( unusedHeader == 0, status = kALAC_ParamError; goto Exit; );

				// read the 1-bit "partial frame" flag, 2-bit "shift-off" flag & 1-bit "escape" flag
				headerByte = (uint8_t) BitBufferRead( bits, 4 );
				
				partialFrame = headerByte >> 3;
				
				bytesShifted = (headerByte >> 1) & 0x3u;
				RequireAction( bytesShifted != 3, status = kALAC_ParamError; goto Exit; );

				shift = bytesShifted * 8;

				escapeFlag = headerByte & 0x1;

				chanBits = mConfig.bitDepth - (bytesShifted * 8) + 1;
				
				// check for partial frame length to override requested numSamples
				if ( partialFrame != 0 )
				{
					numSamples  = BitBufferRead( bits, 16 ) << 16;
					numSamples |= BitBufferRead( bits, 16 );
				}

				if ( escapeFlag == 0 )
				{
					// compressed frame, read rest of parameters
					mixBits		= (uint8_t) BitBufferRead( bits, 8 );
					mixRes		= (int8_t) BitBufferRead( bits, 8 );

					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					modeU		= headerByte >> 4;
					denShiftU	= headerByte & 0xfu;
					
					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					pbFactorU	= headerByte >> 5;
					numU		= headerByte & 0x1fu;
					for ( i = 0; i < numU; i++ )
						coefsU[i] = (int16_t) BitBufferRead( bits, 16 );

					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					modeV		= headerByte >> 4;
					denShiftV	= headerByte & 0xfu;
					
					headerByte	= (uint8_t) BitBufferRead( bits, 8 );
					pbFactorV	= headerByte >> 5;
					numV		= headerByte & 0x1fu;
					for ( i = 0; i < numV; i++ )
						coefsV[i] = (int16_t) BitBufferRead( bits, 16 );

					// if shift active, skip the interleaved shifted values but remember where they start
					if ( bytesShifted != 0 )
					{
						shiftBits = *bits;
						BitBufferAdvance( bits, (bytesShifted * 8) * 2 * numSamples );
					}

					// decompress and run predictor for "left" channel
					set_ag_params( &agParams, mConfig.mb, (pb * pbFactorU) / 4, mConfig.kb, numSamples, numSamples, mConfig.maxRun );
					status = dyn_decomp( &agParams, bits, mPredictor, numSamples, chanBits, &bits1 );
					RequireNoErr( status, goto Exit; );

					if ( modeU == 0 )
					{
						unpc_block( mPredictor, mMixBufferU, numSamples, &coefsU[0], numU, chanBits, denShiftU );
					}
					else
					{
						// the special "numActive == 31" mode can be done in-place
						unpc_block( mPredictor, mPredictor, numSamples, nil, 31, chanBits, 0 );
						unpc_block( mPredictor, mMixBufferU, numSamples, &coefsU[0], numU, chanBits, denShiftU );
					}

					// decompress and run predictor for "right" channel
					set_ag_params( &agParams, mConfig.mb, (pb * pbFactorV) / 4, mConfig.kb, numSamples, numSamples, mConfig.maxRun );
					status = dyn_decomp( &agParams, bits, mPredictor, numSamples, chanBits, &bits2 );
					RequireNoErr( status, goto Exit; );

					if ( modeV == 0 )
					{
						unpc_block( mPredictor, mMixBufferV, numSamples, &coefsV[0], numV, chanBits, denShiftV );
					}
					else
					{
						// the special "numActive == 31" mode can be done in-place
						unpc_block( mPredictor, mPredictor, numSamples, nil, 31, chanBits, 0 );
						unpc_block( mPredictor, mMixBufferV, numSamples, &coefsV[0], numV, chanBits, denShiftV );
					}
				}
				else
				{
					//Assert( bytesShifted == 0 );

					// uncompressed frame, copy data into the mix buffers to use common output code
					chanBits = mConfig.bitDepth;
					shift = 32 - chanBits;
					if ( chanBits <= 16 )
					{
						for ( i = 0; i < numSamples; i++ )
						{
							val = (int32_t) BitBufferRead( bits, (uint8_t) chanBits );
							val = (val << shift) >> shift;
							mMixBufferU[i] = val;

							val = (int32_t) BitBufferRead( bits, (uint8_t) chanBits );
							val = (val << shift) >> shift;
							mMixBufferV[i] = val;
						}
					}
					else
					{
						// BitBufferRead() can't read more than 16 bits at a time so break up the reads
						extraBits = chanBits - 16;
						for ( i = 0; i < numSamples; i++ )
						{
							val = (int32_t) BitBufferRead( bits, 16 );
							val = (val << 16) >> shift;
							mMixBufferU[i] = val | BitBufferRead( bits, (uint8_t)extraBits );

							val = (int32_t) BitBufferRead( bits, 16 );
							val = (val << 16) >> shift;
							mMixBufferV[i] = val | BitBufferRead( bits, (uint8_t)extraBits );
						}
					}

					bits1 = chanBits * numSamples;
					bits2 = chanBits * numSamples;
					mixBits = mixRes = 0;
					bytesShifted = 0;
				}

				// now read the shifted values into the shift buffer
				if ( bytesShifted != 0 )
				{
					shift = bytesShifted * 8;
					//Assert( shift <= 16 );

					for ( i = 0; i < (numSamples * 2); i += 2 )
					{
						mShiftBuffer[i + 0] = (uint16_t) BitBufferRead( &shiftBits, (uint8_t) shift );
						mShiftBuffer[i + 1] = (uint16_t) BitBufferRead( &shiftBits, (uint8_t) shift );
					}
				}

				// un-mix the data and convert to output format
				// - note that mixRes = 0 means just interleave so we use that path for uncompressed frames

				//printf("%d\n", bytesShifted);

				hipMemcpy(d_u + (X * mConfig.frameLength), mMixBufferU, mConfig.frameLength * sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_v + (X * mConfig.frameLength), mMixBufferV, mConfig.frameLength * sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_shiftUV + (X * mConfig.frameLength * 2), mShiftBuffer, mConfig.frameLength * sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_numSamples + X, &numSamples, sizeof(int32_t), hipMemcpyHostToDevice);
				hipMemcpy(d_mixRes + X, &mixRes, sizeof(int8_t), hipMemcpyHostToDevice);
				hipMemcpy(d_mixBits + X, &mixBits, sizeof(uint8_t), hipMemcpyHostToDevice);

				total_numSamples += numSamples;

				channelIndex += 2;
				*outNumSamples = numSamples;
				break;
			}

			case ID_CCE:
			case ID_PCE:
			{
				// unsupported element, bail
				//AssertNoErr( tag );
				status = kALAC_ParamError;
				break;
			}

			case ID_DSE:
			{
				// data stream element -- parse but ignore
				status = this->DataStreamElement( bits );
				break;
			}
			
			case ID_FIL:
			{
				// fill element -- parse but ignore
				status = this->FillElement( bits );
				break;
			}

			case ID_END:
			{
				// frame end, all done so byte align the frame and check for overruns
				BitBufferByteAlign( bits, false );
				//Assert( bits->cur == bits->end );
				goto Exit;
			}
		}

#if ! DEBUG
		// if we've decoded all of our channels, bail (but not in debug b/c we want to know if we're seeing bad bits)
		// - this also protects us if the config does not match the bitstream or crap data bits follow the audio bits
		if ( channelIndex >= numChannels )
			break;
#endif
	}

NoMoreChannels:

	// if we get here and haven't decoded all of the requested channels, fill the remaining channels with zeros
	for ( ; channelIndex < numChannels; channelIndex++ )
	{
		printf("comes here also!");
		/*switch ( mConfig.bitDepth )
		{
			case 16:
			{
				int16_t *	fill16 = &((int16_t *)sampleBuffer)[channelIndex];
				Zero16( fill16, numSamples, numChannels );
				break;
			}
			case 24:
			{
				uint8_t *	fill24 = (uint8_t *)sampleBuffer + (channelIndex * 3);
				Zero24( fill24, numSamples, numChannels );
				break;
			}
			case 32:
			{
				int32_t *	fill32 = &((int32_t *)sampleBuffer)[channelIndex];
				Zero32( fill32, numSamples, numChannels );
				break;
			}
		}*/
	}

Exit:
	return status;
}

#if PRAGMA_MARK
#pragma mark -
#endif

/*
	FillElement()
	- they're just filler so we don't need 'em
*/
int32_t ALACDecoder::FillElement( BitBuffer * bits )
{
	int16_t		count;
	
	// 4-bit count or (4-bit + 8-bit count) if 4-bit count == 15
	// - plus this weird -1 thing I still don't fully understand
	count = BitBufferReadSmall( bits, 4 );
	if ( count == 15 )
		count += (int16_t) BitBufferReadSmall( bits, 8 ) - 1;

	BitBufferAdvance( bits, count * 8 );

	RequireAction( bits->cur <= bits->end, return kALAC_ParamError; );

	return ALAC_noErr;	
}

/*
	DataStreamElement()
	- we don't care about data stream elements so just skip them
*/
int32_t ALACDecoder::DataStreamElement( BitBuffer * bits )
{
	uint8_t		element_instance_tag;
	int32_t		data_byte_align_flag;
	uint16_t		count;
	
	// the tag associates this data stream element with a given audio element
	element_instance_tag = BitBufferReadSmall( bits, 4 );
	
	data_byte_align_flag = BitBufferReadOne( bits );

	// 8-bit count or (8-bit + 8-bit count) if 8-bit count == 255
	count = BitBufferReadSmall( bits, 8 );
	if ( count == 255 )
		count += BitBufferReadSmall( bits, 8 );

	// the align flag means the bitstream should be byte-aligned before reading the following data bytes
	if ( data_byte_align_flag )
		BitBufferByteAlign( bits, false );

	// skip the data bytes
	BitBufferAdvance( bits, count * 8 );

	RequireAction( bits->cur <= bits->end, return kALAC_ParamError; );

	return ALAC_noErr;
}

/*
	ZeroN()
	- helper routines to clear out output channel buffers when decoding fewer channels than requested
*/
static void Zero16( int16_t * buffer, uint32_t numItems, uint32_t stride )
{
	if ( stride == 1 )
	{
		memset( buffer, 0, numItems * sizeof(int16_t) );
	}
	else
	{
		for ( uint32_t index = 0; index < (numItems * stride); index += stride )
			buffer[index] = 0;
	}
}

static void Zero24( uint8_t * buffer, uint32_t numItems, uint32_t stride )
{
	if ( stride == 1 )
	{
		memset( buffer, 0, numItems * 3 );
	}
	else
	{
		for ( uint32_t index = 0; index < (numItems * stride * 3); index += (stride * 3) )
		{
			buffer[index + 0] = 0;
			buffer[index + 1] = 0;
			buffer[index + 2] = 0;
		}
	}
}

static void Zero32( int32_t * buffer, uint32_t numItems, uint32_t stride )
{
	if ( stride == 1 )
	{
		memset( buffer, 0, numItems * sizeof(int32_t) );
	}
	else
	{
		for ( uint32_t index = 0; index < (numItems * stride); index += stride )
			buffer[index] = 0;
	}
}
