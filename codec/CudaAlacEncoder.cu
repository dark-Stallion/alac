#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

#include "CudaAlacEncoder.cuh"

#define SIZE 1024
#define DENSHIFT_DEFAULT 9
#define AINIT 38
#define BINIT (-29)
#define CINIT (-2)


__global__ void call_kALACSearch(int16_t * mCoefsU, int16_t * mCoefsV, int32_t kALACMaxCoefs)
{
	int x = blockIdx.x;
	int y = threadIdx.x;

	int index = x * 16 * 16 + y * 16;
	int32_t		k;
	int32_t		den = 1 << DENSHIFT_DEFAULT;

	mCoefsU[index + 0] = (AINIT * den) >> 4;
	mCoefsU[index + 1] = (BINIT * den) >> 4;
	mCoefsU[index + 2] = (CINIT * den) >> 4;

	mCoefsV[index + 0] = (AINIT * den) >> 4;
	mCoefsV[index + 1] = (BINIT * den) >> 4;
	mCoefsV[index + 2] = (CINIT * den) >> 4;

	for (k = 3; k < kALACMaxCoefs; k++)
	{
		mCoefsU[index + k] = 0;
		mCoefsV[index + k] = 0;
	}
}

void kALACSearch(void  *p1, void *p2, int32_t numPairs, int32_t mNumChannels, int32_t kALACMaxSearches){


	int16_t *d_mCoefsU, *d_mCoefsV;

	hipMalloc(&d_mCoefsU, sizeof(int16_t) * 8 * 16 * 16);
	hipMalloc(&d_mCoefsV, sizeof(int16_t) * 8 * 16 * 16);

	hipMemcpy(d_mCoefsU, p1, sizeof(int16_t) * 8 * 16 * 16, hipMemcpyHostToDevice);
	hipMemcpy(d_mCoefsV, p2, sizeof(int16_t) * 8 * 16 * 16, hipMemcpyHostToDevice);

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	call_kALACSearch << < mNumChannels, kALACMaxSearches >> >(d_mCoefsU, d_mCoefsV, numPairs);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("GPU Time elapsed: %f ms\n", elapsedTime);

	hipMemcpy(p1, d_mCoefsU, sizeof(int16_t) * 8 * 16 * 16, hipMemcpyDeviceToHost);
	hipMemcpy(p2, d_mCoefsV, sizeof(int16_t) * 8 * 16 * 16, hipMemcpyDeviceToHost);

	hipFree(d_mCoefsU);
	hipFree(d_mCoefsV);
}
